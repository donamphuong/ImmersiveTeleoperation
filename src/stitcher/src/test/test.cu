#include "hip/hip_runtime.h"
#include "test.h"

__global__
void copy(pixel *src, pixel *dst, pixel *mask, pixel *dst_mask,
          const int src_width,
          const Point mask_tl, const Rect dst_roi,
          const int dst_width, const int mask_width, const int mask_height,
          const int dst_mask_width) {
  //2D index of current thread
  int x_index = blockIdx.x * blockDim.x + threadIdx.x;
  int y_index = blockIdx.y * blockDim.y + threadIdx.y;

  //only valid threads (threads that copy pixel which are inside mask) perform memory I/O
  //location of top left corner of image in the canvas
  int dx = mask_tl.x - dst_roi.x;
  int dy = mask_tl.y - dst_roi.y;

  if (x_index < dx + mask_width && y_index > dy - mask_height) {
    // for (int y = 0; y < src_height; ++y) {
      //pixel location = row * row_size + col
      int src_pixel = y_index * src_width + x_index;
      int dst_pixel = (dy + y_index) * dst_width + (dx + x_index);
      int mask_pixel = y_index * mask_width + x_index;
      int dst_mask_pixel = (dy + y_index) * dst_mask_width + (dx + x_index);
  
      // for (int x = 0; x < src_width; ++x)
      // {
          if (mask[mask_pixel])
              dst[dst_pixel] = src[src_pixel];
          dst_mask[dst_mask_pixel] |= mask[mask_pixel];
      // }
    // }  
  }
}

void testCopy(const Mat &src, const Mat &dst, const Mat &mask, const Mat &dst_mask, const Point &tl, const Rect &dst_roi) {
  const int srcBytes = src.step * src.rows;
  const int dstBytes = dst.step * dst.rows;
  const int maskBytes = mask.step * mask.rows;
  const int dstMaskBytes = dst_mask.step * dst_mask.rows;

  pixel *d_src, *d_dst, *d_mask, *d_dst_mask;

  //Allocate device memory
  hipMalloc<pixel>(&d_src, srcBytes);
  hipMalloc<pixel>(&d_dst, dstBytes);
  hipMalloc<pixel>(&d_mask, maskBytes);
  hipMalloc<pixel>(&d_dst_mask, dstMaskBytes);

  //Copy data from OpenCV input image to device memory
  hipMemcpy(d_src, src.ptr(), srcBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_mask, mask.ptr(), maskBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_dst_mask, dst_mask.ptr(), dstMaskBytes, hipMemcpyHostToDevice);

  //Sprcify a reasonable block size
  const dim3 block(16, 16);

  //Calculate frid size to cover the whole iamge
  const dim3 grid((dst.cols + block.x - 1)/block.x, (dst.rows + block.y - 1)/block.y);

  copy<<<grid, block>>>(d_src, d_dst, d_mask, d_dst_mask, src.cols, tl, dst_roi, dst.cols, mask.cols, mask.rows, dst_mask.cols);
}


void remapCuda(pixel *src, pixel *dst, UMat UXMap, UMat UYMat) {

}