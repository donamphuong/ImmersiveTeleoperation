#include "hip/hip_runtime.h"
#include "test.h"

__global__
void add(int n, float *x, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

void test() {
  int N = 1 << 20;
  float *x, *y;

  //Allocate Unified memory
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  //Initialise x and y arrays
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  //Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  clock_t start = clock();
  add<<<numBlocks, blockSize>>>(N, x, y);
  cout << "time: " << (clock() - start) / (double) CLOCKS_PER_SEC << endl;


  hipDeviceSynchronize();

  // Free memory
  hipFree(x);
  hipFree(y);
}
