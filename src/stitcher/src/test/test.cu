#include "hip/hip_runtime.h"
#include "test.h"

// #type def unsigned char pixel

__global__
void add(int n, float *x, float *y) {
  //index of the current thread
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  //stride is the total number of threads in the grid
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

__global__
void copy(pixel *src, pixel *dst, pixel *mask, pixel *dst_mask,
          const int src_width,
          const Point mask_tl, const Rect dst_roi,
          const int dst_width, const int mask_width, const int mask_height,
          const int dst_mask_width) {
  //2D index of current thread
  int x_index = blockIdx.x * blockDim.x + threadIdx.x;
  int y_index = blockIdx.y * blockDim.y + threadIdx.y;

  //only valid threads (threads that copy pixel which are inside mask) perform memory I/O
  //location of top left corner of image in the canvas
  int dx = mask_tl.x - dst_roi.x;
  int dy = mask_tl.y - dst_roi.y;

  if (x_index < dx + mask_width && y_index > dy - mask_height) {
    // for (int y = 0; y < src_height; ++y) {
      //pixel location = row * row_size + col
      int src_pixel = y_index * src_width + x_index;
      int dst_pixel = (dy + y_index) * dst_width + (dx + x_index);
      int mask_pixel = y_index * mask_width + x_index;
      int dst_mask_pixel = (dy + y_index) * dst_mask_width + (dx + x_index);
  
      // for (int x = 0; x < src_width; ++x)
      // {
          if (mask[mask_pixel])
              dst[dst_pixel] = src[src_pixel];
          dst_mask[dst_mask_pixel] |= mask[mask_pixel];
      // }
    // }  
  }
}

void testCopy(const Mat &src, const Mat &dst, const Mat &mask, const Mat &dst_mask, const Point &tl, const Rect &dst_roi) {
  const int srcBytes = src.step * src.rows;
  const int dstBytes = dst.step * dst.rows;
  const int maskBytes = mask.step * mask.rows;
  const int dstMaskBytes = dst_mask.step * dst_mask.rows;

  pixel *d_src, *d_dst, *d_mask, *d_dst_mask;

  //Allocate device memory
  hipMalloc<pixel>(&d_src, srcBytes);
  hipMalloc<pixel>(&d_dst, dstBytes);
  hipMalloc<pixel>(&d_mask, maskBytes);
  hipMalloc<pixel>(&d_dst_mask, dstMaskBytes);

  //Copy data from OpenCV input image to device memory
  hipMemcpy(d_src, src.ptr(), srcBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_mask, mask.ptr(), maskBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_dst_mask, dst_mask.ptr(), dstMaskBytes, hipMemcpyHostToDevice);

  //Sprcify a reasonable block size
  const dim3 block(16, 16);

  //Calculate frid size to cover the whole iamge
  const dim3 grid((dst.cols + block.x - 1)/block.x, (dst.rows + block.y - 1)/block.y);

  copy<<<grid, block>>>(d_src, d_dst, d_mask, d_dst_mask, src.cols, tl, dst_roi, dst.cols, mask.cols, mask.rows, dst_mask.cols);
}
 
void test() {
  int N = 1 << 20;
  float *x, *y;

  //Allocate Unified memory
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  //Initialise x and y arrays
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  //Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  clock_t start = clock();
  add<<<numBlocks, blockSize>>>(N, x, y);
  cout << "time: " << (clock() - start) / (double) CLOCKS_PER_SEC << endl;


  hipDeviceSynchronize();

  // Free memory
  hipFree(x);
  hipFree(y);
}

__global__ 
void resizeCudaKernel( unsigned char* input,
  unsigned char* output,
  const int outputWidth,
  const int outputHeight,
  const int inputWidthStep,
  const int outputWidthStep,
  const float pixelGroupSizeX,
  const float pixelGroupSizeY,
  const int inputChannels)
{
//2D Index of current thread
const int outputXIndex = blockIdx.x * blockDim.x + threadIdx.x;
const int outputYIndex = blockIdx.y * blockDim.y + threadIdx.y;

//Only valid threads perform memory I/O
if((outputXIndex<outputWidth) && (outputYIndex<outputHeight))
{
  // Starting location of current pixel in output
  int output_tid  = outputYIndex * outputWidthStep + (outputXIndex * inputChannels);

  // Compute the size of the area of pixels to be resized to a single pixel
  const float pixelGroupArea = pixelGroupSizeX * pixelGroupSizeY;

  // Compute the pixel group area in the input image
  const int intputXIndexStart = int(outputXIndex * pixelGroupSizeX);
  const int intputXIndexEnd = int(intputXIndexStart + pixelGroupSizeX);
  const float intputYIndexStart = int(outputYIndex * pixelGroupSizeY);
  const float intputYIndexEnd = int(intputYIndexStart + pixelGroupSizeY);

  if(inputChannels==1) { // grayscale image
    float channelSum = 0;
    for(int intputYIndex=intputYIndexStart; intputYIndex<intputYIndexEnd; ++intputYIndex) {
      for(int intputXIndex=intputXIndexStart; intputXIndex<intputXIndexEnd; ++intputXIndex) {
        int input_tid = intputYIndex * inputWidthStep + intputXIndex;
        channelSum += input[input_tid];
      }
    }
    output[output_tid] = static_cast<unsigned char>(channelSum / pixelGroupArea);
  } else if(inputChannels==3) { // RGB image
    float channel1stSum = 0;
    float channel2stSum = 0;
    float channel3stSum = 0;
    for(int intputYIndex=intputYIndexStart; intputYIndex<intputYIndexEnd; ++intputYIndex) {
      for(int intputXIndex=intputXIndexStart; intputXIndex<intputXIndexEnd; ++intputXIndex) {
        // Starting location of current pixel in input
        int input_tid = intputYIndex * inputWidthStep + intputXIndex * inputChannels;
        channel1stSum += input[input_tid];
        channel2stSum += input[input_tid+1];
        channel3stSum += input[input_tid+2];
      }
    }
    output[output_tid] = static_cast<unsigned char>(channel1stSum / pixelGroupArea);
    output[output_tid+1] = static_cast<unsigned char>(channel2stSum / pixelGroupArea);
    output[output_tid+2] = static_cast<unsigned char>(channel3stSum / pixelGroupArea);
  } else { 
  }
}
}

void downscaleCuda(const cv::Mat& input, cv::Mat& output)
{
	//Calculate total number of bytes of input and output image
	const int inputBytes = input.step * input.rows;
	const int outputBytes = output.step * output.rows;
 
	unsigned char *d_input, *d_output;
 
	//Allocate device memory
	hipMalloc<unsigned char>(&d_input,inputBytes);
	hipMalloc<unsigned char>(&d_output,outputBytes);
 
  clock_t start = clock();
	//Copy data from OpenCV input image to device memory
	hipMemcpy(d_input,input.ptr(),inputBytes,hipMemcpyHostToDevice);
  double duration = (clock() - start) / (double) CLOCKS_PER_SEC;
	cout << "Copying memory takes " << duration << " secs" << endl;

  start = clock();

	//Specify a reasonable block size
	const dim3 block(16,16);
 
	//Calculate grid size to cover the whole image
	const dim3 grid((output.cols + block.x - 1)/block.x, (output.rows + block.y - 1)/block.y);
 
	// Calculate how many pixels in the input image will be merged into one pixel in the output image
	const float pixelGroupSizeY = float(input.rows) / float(output.rows);
	const float pixelGroupSizeX = float(input.cols) / float(output.cols);
 
	//Launch the size conversion kernel
	resizeCudaKernel<<<grid,block>>>(d_input,d_output,output.cols,output.rows,input.step,output.step, pixelGroupSizeX, pixelGroupSizeY, input.channels());
 
	duration = (clock() - start) / (double) CLOCKS_PER_SEC;
	cout << "OpenCv Gpu code ran in:" << duration << " secs" << endl;
 
	//Synchronize to check for any kernel launch errors
	hipDeviceSynchronize();
 
	//Copy back data from destination device meory to OpenCV output image
	hipMemcpy(output.ptr(),d_output,outputBytes,hipMemcpyDeviceToHost);
 
	//Free the device memory
	hipFree(d_input);
	hipFree(d_output);
	//hipDeviceReset(),"CUDA Device Reset Failed");
}
