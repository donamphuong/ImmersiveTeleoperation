#include "hip/hip_runtime.h"
#include "test.h"

__global__
void add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++) {
    y[i] = x[i] + y[i];
  }
}

void test() {
  int N = 1 << 20;
  float *x, *y;

  //Allocate Unified memory
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  //Initialise x and y arrays
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  //Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

  hipDeviceSynchronize();

  // Free memory
  hipFree(x);
  hipFree(y);
}
